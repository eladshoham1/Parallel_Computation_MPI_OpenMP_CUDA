#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
using namespace std;

#include "cudaFunctions.h"

__global__ void calculateHistogram(int* numbers, int* histogram, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int sharedHistogram[N];
    
    sharedHistogram[threadIdx.x] = 0;
    __syncthreads();

    if (id < size)
        atomicAdd(&(sharedHistogram[numbers[id]]), 1);
    __syncthreads();

    atomicAdd(&histogram[threadIdx.x], sharedHistogram[threadIdx.x]);
}

int checkStatus(hipError_t cudaStatus, int* numbers, int* histogram, string err)
{
    if (cudaStatus != hipSuccess)
    {
        cout << err << endl;

        hipFree(numbers);
        hipFree(histogram);

        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}

int calculateHistogramCuda(int* numbers, int* histogram, int size)
{
    int *devNumbers = 0, *devHistogram = 0;
    int threadsPerBlock = N;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&devNumbers, size * sizeof(int));
    if (checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda malloc failed!") == EXIT_FAILURE)
        return EXIT_FAILURE;

    cudaStatus = hipMalloc((void**)&devHistogram, threadsPerBlock * sizeof(int));
    if (checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda malloc failed!") == EXIT_FAILURE)
        return EXIT_FAILURE;

    cudaStatus = hipMemcpy(devNumbers, numbers, size, hipMemcpyHostToDevice);
    if (checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda memcpy failed!") == EXIT_FAILURE)
        return EXIT_FAILURE;

    cudaStatus = hipMemset(devHistogram, 0, N * sizeof(int));
    if (checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda memset failed!") == EXIT_FAILURE)
        return EXIT_FAILURE;

    calculateHistogram<<<blocksPerGrid, threadsPerBlock>>>(devNumbers, devHistogram, size);
    cudaStatus = hipDeviceSynchronize();
    if (checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda kernel failed!") == EXIT_FAILURE)
        return EXIT_FAILURE;

    cudaStatus = hipMemcpy(histogram, devHistogram, threadsPerBlock, hipMemcpyDeviceToHost);
    if (checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda memcpy failed!") == EXIT_FAILURE)
        return EXIT_FAILURE;

    cudaStatus = hipFree(devNumbers);
    if (checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda free failed!") == EXIT_FAILURE)
        return EXIT_FAILURE;

    cudaStatus = hipFree(devHistogram);
    if (checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda free failed!") == EXIT_FAILURE)
        return EXIT_FAILURE;

    return EXIT_SUCCESS;
}