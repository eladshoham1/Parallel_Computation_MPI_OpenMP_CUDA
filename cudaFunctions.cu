#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include "myProto.h"

enum size { THREADS_PER_BLOCK = 256 };

__global__ void myKernel(int* numbers, int* tempCounters, int size)
{
  //__shared__ int *counters;
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < size)
      tempCounters[i]++;
}

int computeOnGPU(int* numbers, int* tempCounters, int size)
{
  hipError_t err = hipSuccess;
  int *numbersOnGpu, blocksPerGrid = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  err = hipMalloc((void **)&numbersOnGpu, size * sizeof(int));
  if (err != hipSuccess) {
      fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipMemcpy(numbersOnGpu, numbers, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
      fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  myKernel<<<blocksPerGrid, THREADS_PER_BLOCK>>>(numbers, tempCounters, size);
  err = hipGetLastError();
  if (err != hipSuccess) {
      fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  err = hipMemcpy(numbers, numbersOnGpu, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
      fprintf(stderr, "Failed to copy result array from device to host -%s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  if (hipFree(numbersOnGpu) != hipSuccess) {
      fprintf(stderr, "Failed to free device data - %s\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  return EXIT_SUCCESS;
}