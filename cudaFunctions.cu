#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "cudaFunctions.h"

enum size { THREADS_PER_BLOCK = 256 };

__global__ void fillHistogramByZero(int* histogram, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id >= size)
        return;

    histogram[id] = 0;
}

__global__ void calculateHistogram(int* numbers, int* histogram, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int sharedHistogram[THREADS_PER_BLOCK];

    if (id >= size)
        return;

    atomicAdd(sharedHistogram + numbers[id], 1);
    __syncthreads();
    
    histogram[numbers[id]] += sharedHistogram[numbers[id]]; 
}

void checkStatus(hipError_t cudaStatus, int* numbers, int* histogram, const char* err)
{
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "%s\n", err);
        free(numbers);
        free(histogram);
        exit(EXIT_FAILURE);
    }
}

int calculateHistogramCuda(int* numbers, int* histogram, int size)
{
    int *devNumbers = 0, *devHistogram = 0;
    int blocksPerGrid = (size + THREADS_PER_BLOCK) / THREADS_PER_BLOCK;
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void **)&devNumbers, size * sizeof(int));
    checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda malloc failed!");

    cudaStatus = hipMalloc((void **)&devHistogram, THREADS_PER_BLOCK * sizeof(int));
    checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda malloc failed!");

    cudaStatus = hipMemcpy(devNumbers, numbers, size, hipMemcpyHostToDevice);
    checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda memcpy failed!");

    fillHistogramByZero<<<blocksPerGrid, THREADS_PER_BLOCK>>>(devHistogram, THREADS_PER_BLOCK);
    cudaStatus = hipDeviceSynchronize();
    checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda kernel failed!");
    
    calculateHistogram<<<blocksPerGrid, THREADS_PER_BLOCK>>>(devNumbers, devHistogram, size);
    cudaStatus = hipDeviceSynchronize();
    checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda kernel failed!");

    cudaStatus = hipMemcpy(histogram, devHistogram, THREADS_PER_BLOCK, hipMemcpyDeviceToHost);
    checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda memcpy failed!");

    cudaStatus = hipFree(devNumbers);
    checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda free failed!");

    cudaStatus = hipFree(devHistogram);
    checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda free failed!");

    return EXIT_SUCCESS;
}