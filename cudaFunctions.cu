#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#include "cudaFunctions.h"

enum size { THREADS_PER_BLOCK = 256 };

__global__ void calculateHistogram(int* numbers, int* histogram, int size)
{
    __shared__ int privateHistogram[THREADS_PER_BLOCK];
    int id = threadIdx.x;

    if (id < size)
        privateHistogram[id]++;
        
    __syncthreads();
}

void checkStatus(hipError_t cudaStatus, int* numbers, std::string err)
{
    if (cudaStatus != hipSuccess)
    {
        delete[] numbers;
        cout << err << endl;
        exit(EXIT_FAILURE);
    }
}

int calculateHistogramCuda(int* numbers, int* histogram, int size)
{
    int *numbersGpu, *histogramGpu;
    int blocksPerGrid = (size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void **)&numbersGpu, size * sizeof(int));
    checkStatus(cudaStatus, numbersGpu, "Cuda malloc failed!");

    cudaStatus = hipMalloc((void **)&histogramGpu, THREADS_PER_BLOCK * sizeof(int));
    checkStatus(cudaStatus, numbersGpu, "Cuda malloc failed!");

    cudaStatus = hipMemcpy(numbersGpu, numbers, size, hipMemcpyHostToDevice);
    checkStatus(cudaStatus, numbersGpu, "Cuda memcpy failed!");

    cudaStatus = hipMemcpy(histogramGpu, histogram, THREADS_PER_BLOCK, hipMemcpyHostToDevice);
    checkStatus(cudaStatus, numbersGpu, "Cuda memcpy failed!");
    
    calculateHistogram<<<blocksPerGrid, THREADS_PER_BLOCK>>>(numbers, histogram, size);
    cudaStatus = hipDeviceSynchronize();
    checkStatus(cudaStatus, numbersGpu, "Cuda kernel failed!");

    cudaStatus = hipMemcpy(histogram, histogramGpu, THREADS_PER_BLOCK, hipMemcpyDeviceToHost);
    checkStatus(cudaStatus, numbersGpu, "Cuda memcpy failed!");

    cudaStatus = hipFree(numbersGpu);
    checkStatus(cudaStatus, numbersGpu, "Cuda free failed!");

    cudaStatus = hipFree(histogramGpu);
    checkStatus(cudaStatus, numbersGpu, "Cuda free failed!");

    return EXIT_SUCCESS;
}