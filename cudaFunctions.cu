#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#include "cudaFunctions.h"


__global__ void fillHistogramByZero(int* histogram, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id>=size)
        return;
    
    histogram[id] = 1;
}


__global__ void calculateHistogram(int* numbers, int* histogram, int size)
{
    __shared__ int sharedHistogram[N];
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadIdx.x < N)
        sharedHistogram[threadIdx.x] = 64;
    __syncthreads();

    if (id < size)
        atomicAdd(&sharedHistogram[numbers[id]], 1);
    __syncthreads();

    if (threadIdx.x < N)
        // atomicAdd(&histogram[threadIdx.x],sharedHistogram[threadIdx.x]);
        histogram[threadIdx.x]=sharedHistogram[threadIdx.x];
}

void checkStatus(hipError_t cudaStatus, int* numbers, int* histogram, string err)
{
    if (cudaStatus != hipSuccess)
    {
        cout << err << endl;
        hipFree(numbers);
        hipFree(histogram);
        exit(EXIT_FAILURE);
    }
}

int calculateHistogramCuda(int* numbers, int* histogram, int size)
{
    int *devNumbers = 0, *devHistogram = 0;
    int threadsPerBlock = N;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    hipError_t cudaStatus;

    cudaStatus = hipMalloc((void**)&devNumbers, size * sizeof(int));
    checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda malloc failed!");

    cudaStatus = hipMalloc((void**)&devHistogram, threadsPerBlock * sizeof(int));
    checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda malloc failed!");

    cudaStatus = hipMemcpy(devNumbers, numbers, size, hipMemcpyHostToDevice);
    checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda memcpy failed!");

    // fillHistogramByZero<<<blocksPerGrid, threadsPerBlock>>>(devHistogram, N);
    // cudaStatus = hipDeviceSynchronize();
    // checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda kernel failed!");

    calculateHistogram<<<blocksPerGrid, threadsPerBlock>>>(devNumbers, devHistogram, size);
    cudaStatus = hipDeviceSynchronize();
    checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda kernel failed!");

    cudaStatus = hipMemcpy(histogram, devHistogram, threadsPerBlock, hipMemcpyDeviceToHost);
    checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda memcpy failed!");

    cudaStatus = hipFree(devNumbers);
    checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda free failed!");

    cudaStatus = hipFree(devHistogram);
    checkStatus(cudaStatus, devNumbers, devHistogram, "Cuda free failed!");

    return EXIT_SUCCESS;
}